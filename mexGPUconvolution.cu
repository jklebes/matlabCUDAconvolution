#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. dataOut=mexFunction(data).
 *
 * Copyright 2012 The MathWorks, Inc.
 */

#include "mex.h"
#include "gpu/mxGPUArray.h"

// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "convolutionFFT2D_common.h"

#define USE_TEXTURE 1
#define POWER_OF_TWO 1

#if (USE_TEXTURE)
#define LOAD_FLOAT(i) tex1Dfetch<float>(texFloat, i)
#define SET_FLOAT_BASE
#else
#define LOAD_FLOAT(i) d_Src[i]
#define SET_FLOAT_BASE
#endif

////////////////////////////////////////////////////////////////////////////////
/// Position convolution kernel center at (0, 0) in the image
////////////////////////////////////////////////////////////////////////////////
__global__ void padKernel_kernel(float *d_Dst, float *d_Src, int fftH, int fftW,
                                 int kernelH, int kernelW, int kernelY,
                                 int kernelX
#if (USE_TEXTURE)
                                 ,
                                 hipTextureObject_t texFloat
#endif
                                 ) {

  const int y = blockDim.y * blockIdx.y + threadIdx.y;
  const int x = blockDim.x * blockIdx.x + threadIdx.x;

  if ( y < kernelH && x < kernelW) {

    int ky = y - kernelY;

    if (ky < 0) {
      ky += fftH;
    }

    int kx = x - kernelX;

    if (kx < 0) {
      kx += fftW;
    }

    d_Dst[ ky * fftW + kx] = LOAD_FLOAT(y * kernelW + x);
  }
}

////////////////////////////////////////////////////////////////////////////////
// Prepare data for "pad to border" addressing mode
////////////////////////////////////////////////////////////////////////////////
__global__ void unpad_kernel(float *d_Dst, float *d_Src,
                                             int fftH, int fftW, int dataH,
                                            int dataW
#if (USE_TEXTURE)
                                            ,
                                            hipTextureObject_t texFloat
#endif
                                            ) {
  const int y = blockDim.y * blockIdx.y + threadIdx.y;
  const int x = blockDim.x * blockIdx.x + threadIdx.x;

  if (y < dataH && x < dataW) {

    d_Dst[y * dataW + x] = LOAD_FLOAT(y * fftW + x);
  }
}

__global__ void padDataClampToBorder_kernel(float *d_Dst, float *d_Src,
                                             int fftH, int fftW, int dataH,
                                            int dataW,  int kernelH, int kernelW,
                                            int kernelY, int kernelX
#if (USE_TEXTURE)
                                            ,
                                            hipTextureObject_t texFloat
#endif
                                            ) {
  const int y = blockDim.y * blockIdx.y + threadIdx.y;
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int borderH = dataH + kernelY;
  const int borderW = dataW + kernelX;

  if (y < fftH && x < fftW) {
    int  dy, dx;

    if (y < dataH) {
      dy = y;
    }

    if (x < dataW) {
      dx = x;
    }

    if (y >= dataH && y < borderH) {
      dy = dataH - 1;
    }

    if (x >= dataW && x < borderW) {
      dx = dataW - 1;
    }

    if (y >= borderH) {
      dy = 0;
    }

    if (x >= borderW) {
      dx = 0;
    }

    d_Dst[y * fftW + x] = LOAD_FLOAT(dy * dataW + dx);
  }
}

////////////////////////////////////////////////////////////////////////////////
// Modulate Fourier image of padded data by Fourier image of padded kernel
// and normalize by FFT size
////////////////////////////////////////////////////////////////////////////////
inline __device__ void mulAndScale(fComplex &a, const fComplex &b,
                                   const float &c) {
  fComplex t = {c * (a.x * b.x - a.y * b.y), c * (a.y * b.x + a.x * b.y)};
  a = t;
}

__global__ void modulateAndNormalize_kernel(fComplex *d_Dst, fComplex *d_Src,
                                            int dataSize, float c) {
  const int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= dataSize) {
    return;
  }

  fComplex a = d_Src[i];
  fComplex b = d_Dst[i];

  mulAndScale(a, b, c);

  d_Dst[i] = a;
}



////////////////////////////////////////////////////////////////////////////////
/// Position convolution kernel center at (0, 0) in the image
////////////////////////////////////////////////////////////////////////////////
extern "C" void padKernel(float *d_Dst, float *d_Src, int fftH, int fftW,
                           int kernelH, int kernelW, int kernelY, int kernelX) {
  assert(d_Src != d_Dst);
  dim3 threads( 8, 4);
  dim3 grid(iDivUp(kernelW, threads.x), iDivUp(kernelH, threads.y));

  SET_FLOAT_BASE;
#if (USE_TEXTURE)
  hipTextureObject_t texFloat;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeLinear;
  texRes.res.linear.devPtr = d_Src;
  texRes.res.linear.sizeInBytes = sizeof(float) * kernelH * kernelW ;
  texRes.res.linear.desc = hipCreateChannelDesc<float>();

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  hipCreateTextureObject(&texFloat, &texRes, &texDescr, NULL);
#endif

  padKernel_kernel<<<grid, threads>>>(d_Dst, d_Src, fftH, fftW, 
                                      kernelH, kernelW, kernelY, kernelX
#if (USE_TEXTURE)
                                      ,
                                      texFloat
#endif
                                      );

#if (USE_TEXTURE)
  hipDestroyTextureObject(texFloat);
#endif
}

////////////////////////////////////////////////////////////////////////////////
// Prepare data for "pad to border" addressing mode
////////////////////////////////////////////////////////////////////////////////
extern "C" void padDataClampToBorder(float *d_Dst, float *d_Src, 
                                     int fftH, int fftW, int dataH, int dataW,
                                     int kernelH, int kernelW,
                                     int kernelY, int kernelX) {
  assert(d_Src != d_Dst);
  dim3 threads(8, 8);
  dim3 grid(iDivUp(fftW, threads.x), iDivUp(fftH, threads.y));

#if (USE_TEXTURE)
  hipTextureObject_t texFloat;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeLinear;
  texRes.res.linear.devPtr = d_Src;
  texRes.res.linear.sizeInBytes = sizeof(float) * dataH * dataW ;
  texRes.res.linear.desc = hipCreateChannelDesc<float>();

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  hipCreateTextureObject(&texFloat, &texRes, &texDescr, NULL);
#endif

  padDataClampToBorder_kernel<<<grid, threads>>>(
      d_Dst, d_Src,  fftH, fftW,  dataH, dataW,  kernelH, kernelW, kernelY, kernelX
#if (USE_TEXTURE)
      ,
      texFloat
#endif
      );

#if (USE_TEXTURE)
  hipDestroyTextureObject(texFloat);
#endif
}


extern "C" void unpad(float *d_Dst, float *d_Src, 
                                     int fftH, int fftW, int dataH, int dataW ) {
  assert(d_Src != d_Dst);
  dim3 threads(8, 8);
  dim3 grid(iDivUp(fftW, threads.x), iDivUp(fftH, threads.y));

#if (USE_TEXTURE)
  hipTextureObject_t texFloat;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeLinear;
  texRes.res.linear.devPtr = d_Src;
  texRes.res.linear.sizeInBytes = sizeof(float) * fftH * fftW ;
  texRes.res.linear.desc = hipCreateChannelDesc<float>();

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  hipCreateTextureObject(&texFloat, &texRes, &texDescr, NULL);
#endif

  unpad_kernel<<<grid, threads>>>(
      d_Dst, d_Src,  fftH, fftW,  dataH, dataW
#if (USE_TEXTURE)
      ,
      texFloat
#endif
      );

#if (USE_TEXTURE)
  hipDestroyTextureObject(texFloat);
#endif
}

////////////////////////////////////////////////////////////////////////////////
// Modulate Fourier image of padded data by Fourier image of padded kernel
// and normalize by FFT size
////////////////////////////////////////////////////////////////////////////////
extern "C" void modulateAndNormalize(fComplex *d_Dst, fComplex *d_Src, 
                                     int fftH, int fftW, int padding) {
  assert(fftW % 2 == 0);
  const int dataSize =  fftH *(fftW / 2 + padding);
  
  modulateAndNormalize_kernel<<<iDivUp(dataSize, 256), 256 >>>(
      d_Dst, d_Src, dataSize, 1.0f / (float)(fftW * fftH));
}



/*
 * Host code
 */


int snapTransformSize(int dataSize) {
  int hiBit;
  unsigned int lowPOT, hiPOT;

  dataSize = iAlignUp(dataSize, 16);

  for (hiBit = 31; hiBit >= 0; hiBit--)
    if (dataSize & (1U << hiBit)) {
      break;
    }

  lowPOT = 1U << hiBit;

  if (lowPOT == (unsigned int)dataSize) {
    return dataSize;
  }

  hiPOT = 1U << (hiBit + 1);

  if (hiPOT <= 1024) {
    return hiPOT;
  } else {
    return iAlignUp(dataSize, 512);
  }
}

void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare all variables.*/
    mxGPUArray const *data;
    mxGPUArray const *kernel;
    //detect dimensions of data

    float *d_Data;
    float *d_Kernel;
    mxGPUArray *PaddedData_c;
    mxGPUArray *PaddedKernel_c;
    float *d_PaddedKernel;
    float *d_PaddedData;
    mxGPUArray *DataSpectrum_c, * KernelSpectrum_c;
    fComplex* d_DataSpectrum, * d_KernelSpectrum;
    hipfftHandle fftPlanFwd, fftPlanInv;
    float *d_dataOut;
    mxGPUArray *dataOut_c;
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.  Must be matlab single type.";
    char const * const errMsg3D = "Invalid input to MEX file.  3D arrays expected.";

    /* Choose a reasonably sized number of threads for the block. */
    int const threadsPerBlock = 256;
    int blocksPerGrid;

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    /* Throw an error if the input is not a GPU array. */
    if ((nrhs!=2) || !(mxIsGPUArray(prhs[0]))|| !(mxIsGPUArray(prhs[1]))) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }

    data = mxGPUCreateFromMxArray(prhs[0]);
    kernel = mxGPUCreateFromMxArray(prhs[1]);

    //check inputs
    if ((mxGPUGetClassID(data) != mxSINGLE_CLASS)||(mxGPUGetClassID(kernel) != mxSINGLE_CLASS)) { //goes with float
        mexErrMsgIdAndTxt(errId, errMsg);
    }
    if ((mxGPUGetNumberOfDimensions(data) != 2)||(mxGPUGetNumberOfDimensions(kernel) != 2) ) { 
        mexErrMsgIdAndTxt(errId, errMsg3D);
    }

    
    //detect input dimensions
    mwSize const * const dimsData = mxGPUGetDimensions(data);
    mwSize const dataW = dimsData[0]; //I want W to be innermost dimension
    mwSize const dataH = dimsData[1];
    mwSize const * const dimsKernel = mxGPUGetDimensions(kernel);
    mwSize const kernelW = dimsKernel[0]; 
    mwSize const kernelH = dimsKernel[1];
    mwSize const kernelX = kernelW/2; 
    mwSize const kernelY = kernelH/2;
    unsigned int const fftW = snapTransformSize(dataW + kernelW - 1);
    unsigned int const fftH = snapTransformSize(dataH + kernelH - 1);
    mwSize const dimsFft[2] = {fftW, fftH};
    mwSize const dimsComplex[2] = { fftW/2+1 , fftH};

    /*
     * Now that we have verified the data type, extract a pointer to the input
     * data on the device.
     */
    d_Data = (float *)(mxGPUGetDataReadOnly(data));
    d_Kernel = (float *)(mxGPUGetDataReadOnly(kernel));
    /* Create GPUArray on device only. */
    PaddedData_c = mxGPUCreateGPUArray(2,
                            dimsFft,
                            mxGPUGetClassID(data),
                            mxGPUGetComplexity(data),
                            MX_GPU_INITIALIZE_VALUES);
    d_PaddedData = (float *)(mxGPUGetData(PaddedData_c));
    PaddedKernel_c = mxGPUCreateGPUArray(2,
                            dimsFft,
                            mxGPUGetClassID(kernel),
                            mxGPUGetComplexity(kernel),
                            MX_GPU_INITIALIZE_VALUES);
    d_PaddedKernel = (float *)(mxGPUGetData(PaddedKernel_c));
    //fourier space complex arrays
    KernelSpectrum_c = mxGPUCreateGPUArray(2,
                            dimsComplex,
                            mxGPUGetClassID(data), 
                            mxCOMPLEX, 
                            MX_GPU_INITIALIZE_VALUES);
    DataSpectrum_c = mxGPUCreateGPUArray(2,
                            dimsComplex,
                            mxGPUGetClassID(data),
                            mxCOMPLEX,
                            MX_GPU_INITIALIZE_VALUES);
    d_KernelSpectrum = (fComplex *)(mxGPUGetData(KernelSpectrum_c));
    d_DataSpectrum = (fComplex *)(mxGPUGetData(DataSpectrum_c));
    dataOut_c = mxGPUCreateGPUArray(2,
                            dimsData,
                            mxGPUGetClassID(data),
                            mxGPUGetComplexity(data),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_dataOut = (float *)(mxGPUGetData(dataOut_c));
    


    //pad data and kernel
    padKernel(d_PaddedKernel, d_Kernel, fftH, fftW, kernelH, kernelW, kernelY,
        kernelX);
    padDataClampToBorder(d_PaddedData, d_Data, fftH, fftW, dataH, dataW,
        kernelH, kernelW, kernelY, kernelX);

    //Fourier transform
    hipfftPlan2d(&fftPlanFwd, fftH, fftW, HIPFFT_R2C);
    hipfftPlan2d(&fftPlanInv, fftH, fftW, HIPFFT_C2R);
    hipfftExecR2C(fftPlanFwd, (hipfftReal*)d_PaddedKernel,
        (hipfftComplex*)d_KernelSpectrum);
    hipfftExecR2C(fftPlanFwd, (hipfftReal*)d_PaddedData,
        (hipfftComplex*)d_DataSpectrum);

    // multiply elementwise in fourier space + normalize
    modulateAndNormalize((fComplex*)d_DataSpectrum, (fComplex*)d_KernelSpectrum,  fftH, fftW, 1);

    //inverse Fourier transform
    hipfftExecC2R(fftPlanInv, (hipfftComplex*)d_DataSpectrum,
        (hipfftReal*)d_PaddedData);

    //unpad result

    unpad(d_dataOut, d_PaddedData, fftH, fftW,dataH, dataW);

    /* Wrap the result up as a MATLAB gpuArray for return. */
//     plhs[0] = mxGPUCreateMxArrayOnGPU(kernel);
//     plhs[1] = mxGPUCreateMxArrayOnGPU(data);
//     mxGPUArray * fftkernel = mxGPUCopyReal(KernelSpectrum_c);
//     plhs[2] = mxGPUCreateMxArrayOnGPU(fftkernel);
//     mxGPUArray * fftkernel_i = mxGPUCopyImag(KernelSpectrum_c);
//     plhs[3] = mxGPUCreateMxArrayOnGPU(fftkernel_i);
//     mxGPUArray * fftconvolved = mxGPUCopyReal(DataSpectrum_c);
//     plhs[4] = mxGPUCreateMxArrayOnGPU(fftconvolved);
//     mxGPUArray * fftconvolved_i = mxGPUCopyImag(DataSpectrum_c);
//     plhs[5] = mxGPUCreateMxArrayOnGPU(fftconvolved_i);
//     plhs[6] = mxGPUCreateMxArrayOnGPU(PaddedKernel_c);
    plhs[0] = mxGPUCreateMxArrayOnGPU(dataOut_c);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    hipfftDestroy(fftPlanFwd);
    hipfftDestroy(fftPlanInv);

    mxGPUDestroyGPUArray(data);
    mxGPUDestroyGPUArray(kernel);
    mxGPUDestroyGPUArray(PaddedKernel_c);
    mxGPUDestroyGPUArray(PaddedData_c);
    mxGPUDestroyGPUArray(KernelSpectrum_c);
    mxGPUDestroyGPUArray(DataSpectrum_c);
}